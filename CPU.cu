#include "hip/hip_runtime.h"
#include <iostream>
#include "SFML/Graphics.hpp"
#include "hip/hip_runtime.h"
#include <chrono>
#include "fstream"

#define N 510
#define size ((N+2)*(N+2))
#define IX(i, j) ((i) + (N+2) * (j))
#define SWAP(x0,x) {float *tmp=x0;x0=x;x=tmp;}

#define threads 1024
#define blocks (size / threads)

#define SOURCE_SIZE 16
#define FORCE_SIZE 8

// Additional setup for SFML
const int WINDOW_WIDTH = 1024;
const int WINDOW_HEIGHT = 1024;

bool mouse_down[3];
int omx = -1, omy = -1;

///////////////////////////////////////// Initialization function for fluid ////////////////////////////////////////////
void initializeFluid(float * u, float * v, float * u_prev, float * v_prev,float * dens, float * dens_prev ) {
    for (int i = 0; i < size; ++i) {
        u[i] = v[i] = u_prev[i] = v_prev[i] = dens[i] = dens_prev[i] = 0.0f;
    }
}

void reset_arrays(float* arr) {
    for (int i = 0; i <size; ++i) {
        arr[i] = 0.0f;
    }
}


///////////////////////////////////////// simulation functions for fluid CPU////////////////////////////////////////////
void add_source(float * x, float * s, float dt ){
    for (int i=0 ; i<size ; i++) x[i] += dt*s[i];
}

void set_bnd (int b, float *x ){

    for ( int i=1 ; i<=N ; i++ ) {
        x[IX(0 ,i)] = (b==1)? -x[IX(1,i)] : x[IX(1,i)];
        x[IX(N+1,i)] = (b==1)? -x[IX(N,i)] : x[IX(N,i)];
        x[IX(i,0 )] = (b==2)? -x[IX(i,1)] : x[IX(i,1)];
        x[IX(i,N+1)] = (b==2)? -x[IX(i,N)] : x[IX(i,N)];
    }
    x[IX(0 ,0 )] = 0.5f*(x[IX(1,0 )]+x[IX(0 ,1)]);
    x[IX(0 ,N+1)] = 0.5f*(x[IX(1,N+1)]+x[IX(0 ,N )]);
    x[IX(N+1,0 )] = 0.5f*(x[IX(N,0 )]+x[IX(N+1,1)]);
    x[IX(N+1,N+1)] = 0.5f*(x[IX(N,N+1)]+x[IX(N+1,N )]);
}

void lin_solve(int b, float *x, float *x0, float a, float c){
    int i, j, n;
    for ( n=0 ; n<20 ; n++ ) {
        for ( i=1 ; i<=N ; i++ ) {
            for (j = 1; j <= N; j++) {
                x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
            }
        }
        set_bnd(b, x);
    }
}

void diffuse (int b, float * x, float * x0, float diff, float dt ){
    float a=dt*diff*N*N;
    lin_solve(b, x, x0, a, (1+4*a));
}

void advect ( int b, float * d, float * d0, float * u, float * v, float dt ){
    int i, j, i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = dt*N;
    for ( i=1 ; i<=N ; i++ ) {
        for ( j=1 ; j<=N ; j++ ) {
            x = i-dt0*u[IX(i,j)];
            y = j-dt0*v[IX(i,j)];

            if (x<0.5f) x=0.5f;
            if (x>N+0.5f) x=N+ 0.5f;

            i0=(int)x;
            i1=i0+1;

            if (y<0.5f) y=0.5f;
            if (y>N+0.5f) y=N+ 0.5f;

            j0=(int)y;
            j1=j0+1;

            s1 = x-i0;
            s0 = 1-s1;
            t1 = y-j0;
            t0 = 1-t1;

            d[IX(i,j)] = s0*(t0*d0[IX(i0,j0)]+t1*d0[IX(i0,j1)]) + s1*(t0*d0[IX(i1,j0)]+t1*d0[IX(i1,j1)]);
        }
    }
    set_bnd (b,d);
}

void project(float * u, float * v, float * p, float * div){
    int i, j;

    for ( i=1 ; i<=N ; i++ ) {
        for (j = 1; j <= N; j++) {
            div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
            p[IX(i, j)] = 0;
        }
    }

    set_bnd ( 0, div );
    set_bnd ( 0, p );
    lin_solve (0, p, div, 1, 4 );

    for ( i=1 ; i<=N ; i++ ) {
        for (j = 1; j <= N; j++) {
            u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
            v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
        }
    }

    set_bnd ( 1, u );
    set_bnd (2, v );
}

void dens_step (float * x, float * x0, float * u, float * v, float diff,float dt ){
    add_source ( x, x0, dt );
    SWAP ( x0, x ); diffuse( 0, x, x0, diff, dt );
    SWAP ( x0, x ); advect( 0, x, x0, u, v, dt );
}

void vel_step (  float * u, float * v, float * u0, float * v0, float visc, float dt ){
    add_source (  u, u0, dt );
    add_source (  v, v0, dt );
    SWAP ( u0, u ); diffuse ( 1, u, u0, visc, dt );
    SWAP ( v0, v ); diffuse ( 2, v, v0, visc, dt );
    project ( u, v, u0, v0 );
    SWAP ( u0, u );
    SWAP ( v0, v );
    advect (  1, u, u0, u0, v0, dt );
    advect (  2, v, v0, u0, v0, dt );
    project ( u, v, u0, v0 );
}

///////////////////////////////////////// UI function for fluid simulation /////////////////////////////////////////////
void get_from_UI(sf::RenderWindow& window, float* d, float* u, float* v, float& diff, float& visc, float force,
                 float source, bool& simulating, bool& clearData) {

    reset_arrays(d);
    reset_arrays(u);
    reset_arrays(v);

    sf::Event event;
    while (window.pollEvent(event)) {
        switch (event.type) {
            case sf::Event::Closed:
                window.close();
                simulating = false;
                break;
            case sf::Event::KeyPressed:
                switch (event.key.code) {
                    case sf::Keyboard::C:
                        clearData = true;
                        break;
                    case sf::Keyboard::V:
                        //dvel = !dvel;
                        break;
                    case sf::Keyboard::A:
                        diff += 0.00001f;
                        break;
                    case sf::Keyboard::Q:
                        diff = std::max(diff - 0.00001f, 0.0f);
                        break;
                    case sf::Keyboard::Z:
                        visc += 0.00001f;
                        break;
                    case sf::Keyboard::S:
                        visc = std::max(visc - 0.000001f, 0.0f);
                        break;
                    default:
                        break;
                }
                break;
            case sf::Event::MouseButtonPressed:
                mouse_down[event.mouseButton.button] = true;
                break;
            case sf::Event::MouseButtonReleased:
                mouse_down[event.mouseButton.button] = false;
                break;
            case sf::Event::MouseMoved:
                int mx = event.mouseMove.x;
                int my = event.mouseMove.y;
                int i = int((mx / float(WINDOW_WIDTH)) * N + 1);
                int j = int((my / float(WINDOW_HEIGHT)) * N + 1);

                if (1 <= i && i <= N && 1 <= j && j <= N) {
                    if (omx >= 0 && omy >= 0 && mouse_down[sf::Mouse::Left]) {
                        for (int x = std::max(i - FORCE_SIZE, 1); x <= std::min(i + FORCE_SIZE / 4, N); ++x) {
                            for (int y = std::max(j - FORCE_SIZE / 4, 1); y <= std::min(j + FORCE_SIZE / 4, N); ++y) {
                                u[IX(x, y)] = force * (mx - omx);
                                v[IX(x, y)] = force * (my - omy);
                            }
                        }
                    }
                    if (mouse_down[sf::Mouse::Right]) {
                        for (int x = std::max(i - SOURCE_SIZE, 1); x <= std::min(i + SOURCE_SIZE, N); ++x) {
                            for (int y = std::max(j - SOURCE_SIZE, 1); y <= std::min(j + SOURCE_SIZE, N); ++y) {
                                d[IX(x, y)] = source;
                            }
                        }
                    }
                }
                omx = mx;
                omy = my;
                break;
        }
    }
}

__global__ void draw_density_kernel(float* dens, int* colors){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = index % (N + 2); // Convert 1D index to 2D i
    int j = index / (N + 2); // Convert 1D index to 2D j

    if (i >= 1 && i <= N && j >= 1 && j <= N) {

        float avg_dens = (dens[IX(i, j)] + dens[IX(i, j + 1)] + dens[IX(i + 1, j)] + dens[IX(i + 1, j + 1)]) / 4;
        int color_intensity = static_cast<int>(avg_dens * 255);
        color_intensity = (color_intensity < 255)? color_intensity : 255;

        colors[IX(i, j)] = color_intensity;

    }
}

void draw_density(sf::RenderWindow& window,sf::VertexArray& quads,sf::Color& color, float* dens, int *colors,  int*colors_d) {
    //calculate density colors
    //draw_density_kernel<<<blocks, threads>>>(dens, colors_d);
    //hipMemcpy(colors, colors_d, size * sizeof(int), hipMemcpyDeviceToHost);

    //give color to the quads
    for (int i = 1; i <= N; ++i) {
        for (int j = 1; j <= N; ++j) {

            float avg_dens = (dens[IX(i, j)] + dens[IX(i, j + 1)] + dens[IX(i + 1, j)] + dens[IX(i + 1, j + 1)]) / 4;
            int color_intensity = static_cast<int>(avg_dens * 255);
            color_intensity = std::min(color_intensity, 255);

            color.r = color_intensity;
            color.b = color_intensity;
            color.g = color_intensity;

            // Calculate the index in the vertex array
            int quadIndex = ((i - 1) * N + (j - 1)) * 4;

            // Set the color for the vertices
            quads[quadIndex + 0].color = color;
            quads[quadIndex + 1].color = color;
            quads[quadIndex + 2].color = color;
            quads[quadIndex + 3].color = color;
        }
    }

    // Draw the entire set of quads with a single draw call
    window.clear();
    window.draw(quads);
    window.display();
}

void initQuads(sf::VertexArray& quads){
    float h_x = static_cast<float>(WINDOW_WIDTH) / (N);
    float h_y = static_cast<float>(WINDOW_HEIGHT) / (N);

    for (int i = 1; i <= N; ++i) {
        for (int j = 1; j <= N; ++j) {
            float ix_h_x = (i - 1) * h_x; // Precompute to use in positions
            float j_h_y = (j - 1) * h_y; // Precompute to use in positions

            int quadIndex = ((i - 1) * N + (j - 1)) * 4;

            // Define the four corners of the rectangle
            quads[quadIndex + 0].position = sf::Vector2f(ix_h_x, j_h_y);
            quads[quadIndex + 1].position = sf::Vector2f(i * h_x, j_h_y);
            quads[quadIndex + 2].position = sf::Vector2f(i * h_x, j * h_y);
            quads[quadIndex + 3].position = sf::Vector2f(ix_h_x, j * h_y);
        }
    }
}
//////////////////////////////////////////////////// main //////////////////////////////////////////////////////////////
int main() {

    std::ofstream myFile("fluid_simulation_CPU.csv");
    if(!myFile.is_open()){
        std::cout<< "failed to open the file." << std::endl;
        return 1;
    }

    sf::RenderWindow window(sf::VideoMode(WINDOW_WIDTH, WINDOW_HEIGHT), "Fluid Simulation");
    sf::VertexArray quads(sf::Quads, 4 * N * N);
    sf::Color color(0, 0, 0);
    initQuads(quads);

    static float u[size], v[size], u_prev[size], v_prev[size];
    static float dens[size], dens_prev[size];
    int *colors = (int*) malloc(size*sizeof(int));

    float *u_d, *v_d, *u_prev_d, *v_prev_d;
    float *dens_d, *dens_prev_d;
    int *colors_d;

// Allocate memory on the device
    hipMalloc((void **)&u_d, size * sizeof(float));
    hipMalloc((void **)&v_d, size * sizeof(float));
    hipMalloc((void **)&u_prev_d, size * sizeof(float));
    hipMalloc((void **)&v_prev_d, size * sizeof(float));
    hipMalloc((void **)&dens_d, size * sizeof(float));
    hipMalloc((void **)&dens_prev_d, size * sizeof(float));
    hipMalloc((void **)&colors_d, size * sizeof(int));

    float source = 1.0f;
    float force = 10.0f;

    float visc = 0.000000f;
    float diff = 0.00000f;
    float dt = 0.05f;

    bool simulating = true;
    bool clearData = true;

    int teller = 0;

    while(simulating){
        while(window.isOpen()) {

            if(clearData){
                initializeFluid(u, v, u_prev, v_prev, dens, dens_prev);
                hipMemcpy(u_d, u, size * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(v_d, v, size * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(dens_d, dens, size * sizeof(float), hipMemcpyHostToDevice);
                clearData = false;
            }

            auto startCPU = std::chrono::high_resolution_clock::now();

            get_from_UI(window, dens_prev, u_prev, v_prev, diff,visc,force, source, simulating, clearData);

            //hipMemcpy(u_prev_d, u_prev, size * sizeof(float), hipMemcpyHostToDevice);
            //hipMemcpy(v_prev_d, v_prev, size * sizeof(float), hipMemcpyHostToDevice);
            //hipMemcpy(dens_prev_d, dens_prev, size * sizeof(float), hipMemcpyHostToDevice);

            vel_step(u, v, u_prev, v_prev, visc, dt);
            dens_step(dens, dens_prev, u, v, diff, dt);

            //hipMemcpy(u_prev, u_prev_d, size * sizeof(float), hipMemcpyDeviceToHost);
            //hipMemcpy(v_prev, v_prev_d, size * sizeof(float), hipMemcpyDeviceToHost);
            //hipMemcpy(dens_prev, dens_prev_d, size * sizeof(float), hipMemcpyDeviceToHost);

            draw_density(window,quads, color,dens, colors, colors_d);

            auto stopCPU = std::chrono::high_resolution_clock::now();
            std::chrono::duration<float, std::milli> time = stopCPU - startCPU;
            //std::cout<<" fps " << 1000/time.count() << std::endl;
            if(teller < 1000){
                myFile << time.count() << "\n";
                teller++;
            }
            if(teller == 1000) std::cout<< "file is complete";

        }
    }

    myFile.close();

    free(colors);

    hipFree(u_d);
    hipFree(v_d);
    hipFree(u_prev_d);
    hipFree(v_prev_d);
    hipFree(dens_d);
    hipFree(dens_prev_d);
    hipFree(colors_d);

    return 0;
}
